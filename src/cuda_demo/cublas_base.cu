#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hipblas.h"

using namespace std;

// C = A * B
int main(int argc, char const *argv[])
{
    int M = 4;  // 矩阵A的行数，矩阵C的行数
    int N = 4;  // 矩阵A的列数，矩阵B的行数
    int K = 4;  // 矩阵B的列数，矩阵C的列数

    // init data buffer
    float *host_a;
    float *host_b;
    float *host_c1;  // 用于接收stream1的result
    float *host_c2;  // 用于接收stream2的result
    hipHostAlloc((void **)&host_a, M*N*sizeof(float), hipHostMallocDefault);
    hipHostAlloc((void **)&host_b, N*K*sizeof(float), hipHostMallocDefault);
    hipHostAlloc((void **)&host_c1, M*K*sizeof(float), hipHostMallocDefault);
    hipHostAlloc((void **)&host_c2, M*K*sizeof(float), hipHostMallocDefault);

    // create data
    for (size_t i = 0; i < M * N; i++)
    {
        host_a[i] = i;
    }

    for (size_t i = 0; i < N * K; i++)
    {
        host_b[i] = i;
    }

    // init device for stream1
    float *dev_a1;
    float *dev_b1;
    float *dev_c1;

    // init device for stream2
    float *dev_a2;
    float *dev_b2;
    float *dev_c2;

    hipMalloc((void **)&dev_a1, M*N*sizeof(float));
    hipMalloc((void **)&dev_b1, N*K*sizeof(float));
    hipMalloc((void **)&dev_c1, M*K*sizeof(float));
    hipMalloc((void **)&dev_a2, M*N*sizeof(float));
    hipMalloc((void **)&dev_b2, N*K*sizeof(float));
    hipMalloc((void **)&dev_c2, M*K*sizeof(float));

    // create streams
    hipStream_t stream1;
    hipStream_t stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    // create handles
    hipblasHandle_t handle1;
    hipblasHandle_t handle2;
    hipblasCreate(&handle1);
    hipblasCreate(&handle2);

    // 为handle设置stream
    hipblasSetStream(handle1, stream1);
    hipblasSetStream(handle2, stream2);

    // 将数据拷入device中
    hipMemcpyAsync(dev_a1, host_a, M*N*sizeof(float), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(dev_b1, host_b, N*K*sizeof(float), hipMemcpyHostToDevice, stream1);

    hipMemcpyAsync(dev_a2, host_a, M*N*sizeof(float), hipMemcpyHostToDevice, stream2);
    hipMemcpyAsync(dev_b2, host_b, N*K*sizeof(float), hipMemcpyHostToDevice, stream2);

    float alpha = 1; 
    float beta = 0;

    // 调用cublas中的gemm函数
    hipblasSgemm(handle1, 
                    HIPBLAS_OP_N,   // 矩阵A是否转置
                    HIPBLAS_OP_N,   // 矩阵B是否转置
                    M, 
                    N, 
                    K, 
                    &alpha,   // 乘积的值
                    dev_a1, 
                    M,        // 数据存储的是连续的地址，M的作用是A每隔几个数据作为一行
                    dev_b1, 
                    N, 
                    &beta, 
                    dev_c1, 
                    M);

    hipblasSgemm(handle2, 
                    HIPBLAS_OP_N,   // 矩阵A是否转置
                    HIPBLAS_OP_N,   // 矩阵B是否转置
                    M, 
                    N, 
                    K, 
                    &alpha,   // 乘积的值
                    dev_a2, 
                    M,        // 数据存储的是连续的地址，M的作用是A每隔几个数据作为一行
                    dev_b2, 
                    N, 
                    &beta, 
                    dev_c2, 
                    M);

    // 将数据拷贝回host
    hipMemcpyAsync(host_c1, dev_c1, M*K*sizeof(float), hipMemcpyDeviceToHost, stream1);
    hipMemcpyAsync(host_c2, dev_c2, M*K*sizeof(float), hipMemcpyDeviceToHost, stream2);
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    // 打印计算结果
    cout << "Result:" << endl;
    for (size_t i = 0; i < M; i++)
    {
        for (size_t j = 0; j < K; j++)
        {
            cout << host_c1[j+i*M] << " ";
            if ((j+1+i*M) % M == 0)
            {
                cout << endl;
            }
        }
    }
    

    // 资源释放
    hipFree(dev_a1);
    hipFree(dev_a2);
    hipFree(dev_b1);
    hipFree(dev_b2);
    hipFree(dev_c1);
    hipFree(dev_c2);

    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c1);
    hipHostFree(host_c2);

    hipblasDestroy(handle1);
    hipblasDestroy(handle2);

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);


    
    return 0;
}

// nvcc cublas_base.cu -o cublas_base.out -lcublas