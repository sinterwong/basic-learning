#include "hip/hip_runtime.h"
#include <stdio.h>

// (A+B)/2=C
#define N (4096 * 4096) // 每个stream执行数据的大小
#define FULL (N * 20)   // 全部数据的大小

__global__ void kernel(int *a, int *b, int *c)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < N)
    {
        c[idx] = (a[idx] + b[idx]) / 2;
    }
}

int main(int argc, char const *argv[])
{
    // 检查设备属性是否支持Stream
    hipDeviceProp_t prop;
    int whichDevice;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if (!prop.deviceOverlap)
    {
        printf("Device will not support overlap!\n");
        return 0;
    }

    // init 计时器 event
    hipEvent_t start, end;
    float elapsedTime;

    // create 计时器
    hipEventCreate(&start);
    hipEventCreate(&end);

    // 声明并创建Stream
    hipStream_t stream;
    hipStreamCreate(&stream);

    // 声明Buffer指针
    int *host_a;
    int *host_b;
    int *host_c;

    int *dev_a;
    int *dev_b;
    int *dev_c;

    // 锁页内存分配
    hipHostAlloc((void **)&host_a, FULL * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void **)&host_b, FULL * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void **)&host_c, FULL * sizeof(int), hipHostMallocDefault);

    // Device 内存分配
    hipMalloc((void **)&dev_a, N * sizeof(int));
    hipMalloc((void **)&dev_b, N * sizeof(int));
    hipMalloc((void **)&dev_c, N * sizeof(int));
    // hipHostAlloc((void **)&dev_a, N * sizeof(int), hipHostMallocDefault);
    // hipHostAlloc((void **)&dev_b, N * sizeof(int), hipHostMallocDefault);
    // hipHostAlloc((void **)&dev_c, N * sizeof(int), hipHostMallocDefault);

    // 为A和B赋值
    for (int i = 0; i < FULL; i++)
    {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    // 运作计时器
    hipEventRecord(start, 0);

    // 分流异步计算
    for (int i = 0; i < FULL; i+=N)
    {
        // 将锁页内存上的数据拷贝到Device上
        hipMemcpyAsync(dev_a, host_a+i, N*sizeof(int), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(dev_b, host_b+i, N*sizeof(int), hipMemcpyHostToDevice, stream);
        
        // <<<gridDim, blockDim, 使用shared_mem大小, stream>>>
        kernel<<<N/256, 256, 0, stream>>>(dev_a, dev_b, dev_c);

        // 将计算结果copy到Host上
        hipMemcpyAsync(host_c+i, dev_c, N*sizeof(int), hipMemcpyDeviceToHost, stream);
    }

    hipStreamSynchronize(stream);

    // 耗时计算
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsedTime, start, end);
    printf("cost time: %3.3f ms\n", elapsedTime);
    
    // 资源释放
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);    
    hipStreamDestroy(stream);


    return 0;
}
