#include <cstddef>
#include <iostream>
#include <array>
#include <hip/hip_runtime.h>
#include <numeric>

#define threadsPerBlock 256

__global__ void arraySum(const float* a, float *ret) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int block_id = blockIdx.x;
  int block_tid = threadIdx.x;
  __shared__ float sData[threadsPerBlock];  // 申请共享数据内存（每个block中的线程共享）
  sData[block_tid] = a[tid];
  __syncthreads();
  for (int i = threadsPerBlock / 2; i > 0; i /= 2) {
    if (block_tid < i) {
      sData[block_tid] = sData[block_tid] + sData[block_tid + i];
    }
    __syncthreads();
  }
  if (block_tid == 0) {
    ret[block_id] = sData[0];
  }
}

int main(int argc, char **argv) {

  // 数据数量需要符合2^n，不够需要补0
  constexpr int numElements = 5120;

  const int blockPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;  // block数量

  std::array<float, numElements> data;
  for (int i = 0; i < data.size(); i ++) {
    data[i] = i;
  }
  // for (auto i : data) {
  //   std::cout << i << ", ";
  // }
  // std::cout << std::endl;

  size_t size = numElements * sizeof(float);

  std::array<float, blockPerGrid> ret; // result on the host
  float *d_data = nullptr;  // data of device
  float *d_ret = nullptr; // result on the device
  hipMalloc((void**)&d_data, size);
  hipMalloc((void**)&d_ret, blockPerGrid * sizeof(float));

  hipMemcpy(d_data, data.begin(), size, hipMemcpyHostToDevice);
  
  arraySum<<<blockPerGrid, threadsPerBlock>>>(d_data, d_ret);
  hipMemcpy(ret.begin(), d_ret, blockPerGrid * sizeof(float), hipMemcpyDeviceToHost);

  int result = std::accumulate(ret.begin(), ret.end(), 0);

  std::cout << "result: " <<  result << std::endl;
  hipFree(d_data);
  hipFree(d_ret);
  return 0;
}
