#include "hip/hip_runtime.h"
#include <stdio.h>

#define N (1024 * 1024)
#define FULL (N * 20)

__global__ void kernel(int *a, int *b, int *c)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < N)
    {
        c[idx] = (a[idx] + b[idx]) / 2;
    }
}

int main(int argc, char const *argv[])
{

    // 检查GPU属性是否支持Stream
    hipDeviceProp_t prop;
    int whichDevice;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if (!prop.deviceOverlap)
    {
        printf("Device will not support overlap!\n");
        return 0;
    }

    // init 计时 event & create 计时器
    hipEvent_t start;
    hipEvent_t end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    float elapsed;

    // init stream & create
    hipStream_t stream1;
    hipStream_t stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    // 声明buffer & 锁页内存分配 & cuda内存分配
    int *host_a;
    int *host_b;
    int *host_c;
    int *dev_a1;
    int *dev_b1;
    int *dev_c1;
    int *dev_a2;
    int *dev_b2;
    int *dev_c2;
    hipHostAlloc((void **)&host_a, FULL * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void **)&host_b, FULL * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void **)&host_c, FULL * sizeof(int), hipHostMallocDefault);
    hipMalloc((void **)&dev_a1, N * sizeof(int));
    hipMalloc((void **)&dev_b1, N * sizeof(int));
    hipMalloc((void **)&dev_c1, N * sizeof(int));
    hipMalloc((void **)&dev_a2, N * sizeof(int));
    hipMalloc((void **)&dev_b2, N * sizeof(int));
    hipMalloc((void **)&dev_c2, N * sizeof(int));

    // A和B赋值
    for (size_t i = 0; i < FULL; i++)
    {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    // 开始计时
    hipEventRecord(start, 0);

    // 多流异步计算
    for (size_t i = 0; i < FULL; i+=(N*2))
    {
        // 将锁页内存拷贝到Device
        hipMemcpyAsync(dev_a1, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(dev_a2, host_a + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream2);

        hipMemcpyAsync(dev_b1, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(dev_b2, host_b + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream2);

        kernel<<<N/256, 256, 0, stream1>>>(dev_a1, dev_b1, dev_c1);
        kernel<<<N/256, 256, 0, stream2>>>(dev_a2, dev_b2, dev_c2);

        // 将计算结果拷贝回host
        hipMemcpyAsync(host_c+i, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1);
        hipMemcpyAsync(host_c+i+N, dev_c2, N * sizeof(int), hipMemcpyDeviceToHost, stream2);
    }
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed, start, end);
    printf("cost time: %3.3f ms\n", elapsed);

    // relased resources 
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipFree(dev_a1);
    hipFree(dev_a2);
    hipFree(dev_b1);
    hipFree(dev_b2);
    hipFree(dev_c1);
    hipFree(dev_c2);
    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);

    return 0;
}
